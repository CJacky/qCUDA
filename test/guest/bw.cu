#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>

#include "common.h"

int main(int argc, char* argv[])
{
	uint8_t *H, *D;
	uint64_t size = 0, err = 0;
	int i;

	printf("0 "); //reg

	for(i=0; i<strlen(argv[1]); i++)
		size = size*10 + (argv[1][i]-'0');

	H = (uint8_t*)malloc(sizeof(uint8_t)*size);
	
	time_begin();
	hipMalloc((void**)&D, sizeof(uint8_t)*size);
	time_end();

	for(i=0; i<size; i++)
	{
		H[i] = i%255;
	}

	time_begin();
	hipMemcpy(D, H, size*sizeof(uint8_t), hipMemcpyHostToDevice);
	time_end();

	for(i=0; i<size; i++)
	{
		H[i]=0;
	}

	printf("0 "); //launch

	time_begin();
	hipMemcpy(H, D, size*sizeof(uint8_t), hipMemcpyDeviceToHost);
	time_end();

	for(i=0; i<size; i++)
	{
		if(H[i] != i%255)
			err++;
	}

	if(err)
		fprintf(stderr, "bw %"PRIu64" error %"PRIu64"\n", size, err);

	free(H);
	
	time_begin();
	hipFree(D);
	time_end();

	return 0;
}
