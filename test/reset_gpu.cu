#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
	int devNum;

	hipGetDeviceCount(&devNum);

	printf("there are %d gpus\n", devNum);
	hipSetDevice(0);
	hipDeviceReset();
	return 0;
}
